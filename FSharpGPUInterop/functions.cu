/*This file is part of FSharpGPU.

	FSharpGPU is free software : you can redistribute it and / or modify
	it under the terms of the GNU General Public License as published by
	the Free Software Foundation, either version 3 of the License, or
	(at your option) any later version.

	FSharpGPU is distributed in the hope that it will be useful,
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
	GNU General Public License for more details.

	You should have received a copy of the GNU General Public License
	along with FSharpGPU.If not, see <http://www.gnu.org/licenses/>.
*/

/* This software contains source code provided by NVIDIA Corporation. */

/* Copyright � 2015 Philip Curzon */

#include "definitions.cuh"
#include "functions.cuh"


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <functional>
#include <algorithm>

/* Create an uninitialised cuda array of length n, where each element has size typeSize */
int createCUDAArray(size_t n, size_t typeSize, void **devPtr)
{
	hipError_t cudaStatus;
	__int32 byteSize = n * typeSize;
	if ((cudaStatus = hipMalloc(devPtr, byteSize)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

/* Free a cuda array */
int freeCUDAArray(void *devPtr)
{
	return hipFree(devPtr);
}

/* Create an uninitialised array of doubles of length n */
int createCUDADoubleArray(size_t n, double **devPtr)
{
	hipError_t cudaStatus;
	size_t byteSize = n * sizeof(double);
	if ((cudaStatus = hipMalloc((void**)devPtr, byteSize)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}


/* Create and initialise array of doubles of length n */
int initialiseCUDADoubleArray(const double *array, const size_t n, double **devPtr)
{
	hipError_t cudaStatus;
	size_t byteSize = n * sizeof(double);
	//if ((cudaStatus = hipMalloc((void**)devPtr, byteSize)) != hipSuccess) return cudaStatus;
	if ((cudaStatus = (hipError_t)createCUDADoubleArray(n, devPtr)) != hipSuccess) return cudaStatus;
	if ((cudaStatus = hipMemcpy(*devPtr, array, byteSize, hipMemcpyHostToDevice)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

/* Retreive the contents of an array of cuda doubles */
int retrieveCUDADoubleArray(double *devPtr, const size_t offset, double dblArray[], const size_t n)
{
	hipError_t cudaStatus;
	size_t byteSize = n * sizeof(double);
	if ((cudaStatus = hipMemcpy(dblArray, devPtr+offset, byteSize, hipMemcpyDeviceToHost)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

/* Create an uninitialised array of doubles of length n */
int createCUDABoolArray(size_t n, __int32 **devPtr)
{
	hipError_t cudaStatus;
	size_t byteSize = n * sizeof(int);
	if ((cudaStatus = hipMalloc((void**)devPtr, byteSize)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

/* Create and initialise array of bools of length n */
int initialiseCUDABoolArray(const __int32 *array, const size_t n, __int32 **devPtr)
{
	hipError_t cudaStatus;
	size_t byteSize = n * sizeof(bool);
	if ((cudaStatus = hipMalloc((void**)devPtr, byteSize)) != hipSuccess) return cudaStatus;
	if ((cudaStatus = hipMemcpy(*devPtr, array, byteSize, hipMemcpyHostToDevice)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

/* Retreive the contents of an array of cuda bools */
int retrieveCUDABoolArray(__int32 *devPtr, const size_t offset, __int32 dblArray[], const size_t n)
{
	hipError_t cudaStatus;
	size_t byteSize = n * sizeof(int);
	if ((cudaStatus = hipMemcpy(dblArray, devPtr + offset, byteSize, hipMemcpyDeviceToHost)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}
