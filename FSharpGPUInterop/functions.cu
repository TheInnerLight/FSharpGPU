/*This file is part of FSharpGPU.

	FSharpGPU is free software : you can redistribute it and / or modify
	it under the terms of the GNU General Public License as published by
	the Free Software Foundation, either version 3 of the License, or
	(at your option) any later version.

	FSharpGPU is distributed in the hope that it will be useful,
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
	GNU General Public License for more details.

	You should have received a copy of the GNU General Public License
	along with FSharpGPU.If not, see <http://www.gnu.org/licenses/>.
*/

/* Copyright � 2015 Philip Curzon */

#include "definitions.cuh"
#include "functions.cuh"


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <functional>
#include <algorithm>

/* Create an uninitialised cuda array of length n, where each element has size typeSize */
int createCUDAArray(int n, int typeSize, void **devPtr)
{
	hipError_t cudaStatus;
	int byteSize = n * typeSize;
	if ((cudaStatus = hipMalloc(devPtr, byteSize)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

/* Free a cuda array */
int freeCUDAArray(void *devPtr)
{
	return hipFree(devPtr);
}

/* Create an uninitialised array of doubles of length n */
int createCUDADoubleArray(int n, double **devPtr)
{
	hipError_t cudaStatus;
	int byteSize = n * sizeof(double);
	if ((cudaStatus = hipMalloc((void**)devPtr, byteSize)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}



/* Create and initialise array of doubles of length n */
int initialiseCUDADoubleArray(const double *array, const int n, double **devPtr)
{
	hipError_t cudaStatus;
	int byteSize = n * sizeof(double);
	if ((cudaStatus = hipMalloc((void**)devPtr, byteSize)) != hipSuccess) return cudaStatus;
	if ((cudaStatus = hipMemcpy(*devPtr, array, byteSize, hipMemcpyHostToDevice)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

/* Retreive the contents of an array of cuda doubles */
int retrieveCUDADoubleArray(double *devPtr, const int offset, double dblArray[], const int n)
{
	hipError_t cudaStatus;
	int byteSize = n * sizeof(double);
	if ((cudaStatus = hipMemcpy(dblArray, devPtr+offset, byteSize, hipMemcpyDeviceToHost)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

int createCUDABoolArray(int n, int **devPtr)
{
	hipError_t cudaStatus;
	int byteSize = n * sizeof(int);
	if ((cudaStatus = hipMalloc((void**)devPtr, byteSize)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

int initialiseCUDABoolArray(const int *array, const int n, int **devPtr)
{
	hipError_t cudaStatus;
	int byteSize = n * sizeof(bool);
	if ((cudaStatus = hipMalloc((void**)devPtr, byteSize)) != hipSuccess) return cudaStatus;
	if ((cudaStatus = hipMemcpy(*devPtr, array, byteSize, hipMemcpyHostToDevice)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

int retrieveCUDABoolArray(int *devPtr, const int offset, int dblArray[], const int n)
{
	hipError_t cudaStatus;
	int byteSize = n * sizeof(int);
	if ((cudaStatus = hipMemcpy(dblArray, devPtr + offset, byteSize, hipMemcpyDeviceToHost)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}
