/*This file is part of FSharpGPU.

	FSharpGPU is free software : you can redistribute it and / or modify
	it under the terms of the GNU General Public License as published by
	the Free Software Foundation, either version 3 of the License, or
	(at your option) any later version.

	FSharpGPU is distributed in the hope that it will be useful,
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
	GNU General Public License for more details.

	You should have received a copy of the GNU General Public License
	along with FSharpGPU.If not, see <http://www.gnu.org/licenses/>.
*/

/* This software contains source code provided by NVIDIA Corporation. */

/* Copyright � 2015 Philip Curzon */

#include "definitions.cuh"
#include "functions.cuh"


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <functional>
#include <algorithm>

/* Create an uninitialised cuda array of length n, where each element has size typeSize */
__int32 createCUDAArray(size_t n, size_t typeSize, void **devPtr)
{
	hipError_t cudaStatus;
	__int32 byteSize = n * typeSize;
	if ((cudaStatus = hipMalloc(devPtr, byteSize)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

/* Free a cuda array */
__int32 freeCUDAArray(void *devPtr)
{
	return hipFree(devPtr);
}

/* Create an uninitialised array of doubles of length n */
__int32 createCUDADoubleArray(size_t n, double **devPtr)
{
	hipError_t cudaStatus;
	__int32 byteSize = n * sizeof(double);
	if ((cudaStatus = hipMalloc((void**)devPtr, byteSize)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}



/* Create and initialise array of doubles of length n */
__int32 initialiseCUDADoubleArray(const double *array, const size_t n, double **devPtr)
{
	hipError_t cudaStatus;
	__int32 byteSize = n * sizeof(double);
	if ((cudaStatus = hipMalloc((void**)devPtr, byteSize)) != hipSuccess) return cudaStatus;
	if ((cudaStatus = hipMemcpy(*devPtr, array, byteSize, hipMemcpyHostToDevice)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

/* Retreive the contents of an array of cuda doubles */
__int32 retrieveCUDADoubleArray(double *devPtr, const size_t offset, double dblArray[], const size_t n)
{
	hipError_t cudaStatus;
	__int32 byteSize = n * sizeof(double);
	if ((cudaStatus = hipMemcpy(dblArray, devPtr+offset, byteSize, hipMemcpyDeviceToHost)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

__int32 createCUDABoolArray(size_t n, __int32 **devPtr)
{
	hipError_t cudaStatus;
	__int32 byteSize = n * sizeof(int);
	if ((cudaStatus = hipMalloc((void**)devPtr, byteSize)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

__int32 initialiseCUDABoolArray(const __int32 *array, const size_t n, __int32 **devPtr)
{
	hipError_t cudaStatus;
	__int32 byteSize = n * sizeof(bool);
	if ((cudaStatus = hipMalloc((void**)devPtr, byteSize)) != hipSuccess) return cudaStatus;
	if ((cudaStatus = hipMemcpy(*devPtr, array, byteSize, hipMemcpyHostToDevice)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}

__int32 retrieveCUDABoolArray(__int32 *devPtr, const size_t offset, __int32 dblArray[], const size_t n)
{
	hipError_t cudaStatus;
	__int32 byteSize = n * sizeof(int);
	if ((cudaStatus = hipMemcpy(dblArray, devPtr + offset, byteSize, hipMemcpyDeviceToHost)) != hipSuccess) return cudaStatus;
	return cudaStatus;
}
