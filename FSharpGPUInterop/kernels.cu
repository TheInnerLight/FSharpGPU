#include "hip/hip_runtime.h"
/*This file is part of FSharpGPU.

FSharpGPU is free software : you can redistribute it and / or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

FSharpGPU is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with FSharpGPU.If not, see <http://www.gnu.org/licenses/>.
*/

/* This software contains source code provided by NVIDIA Corporation. */

/* Copyright � 2015 Philip Curzon */



#include "definitions.cuh"
#include "kernels.cuh"
#include "functions.cuh"
#include "scankernels.cuh"
#include "templated_kernels.cuh"

#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <algorithm>



/* Split the size of the array between threads and blocks */
ThreadBlocks getThreadsAndBlocks(const int n)
{
	ThreadBlocks tb;
	tb.threadCount = std::min(MAX_THREADS, n);
	tb.blockCount = std::min(MAX_BLOCKS, std::max(1, (n + tb.threadCount - 1) / tb.threadCount));
	tb.thrBlockCount = tb.threadCount * tb.blockCount;
	tb.loopCount = std::min(MAX_BLOCKS, std::max(1, (n + tb.thrBlockCount - 1) / tb.thrBlockCount));
	tb.N = n;
	return tb;
}

/* Split the size of the array between threads and blocks */
ThreadBlocks getThreadsAndBlocks32(const int n)
{
	ThreadBlocks tb;
	
	tb.threadCount = std::min(MAX_THREADS, n);
	tb.loopCount = std::min(32, std::max(1, (n + tb.threadCount - 1) / tb.threadCount));
	__int32 thrLoopCount = tb.loopCount * tb.threadCount;
	tb.blockCount = std::min(MAX_BLOCKS, std::max(1, (n + thrLoopCount - 1) / thrLoopCount));
	tb.thrBlockCount = tb.threadCount * tb.blockCount;
	tb.N = n;
	return tb;
}

__device__ void getInputArrayValueForIndexingScheme(double *inputArr, const int inputOffset, const int inputN, int scheme, double *val)
{
	switch (scheme)
	{
	case 0:
		if (((blockIdx.x * blockDim.x + threadIdx.x + inputOffset) >= inputN) || ((blockIdx.x * blockDim.x + threadIdx.x + inputOffset) < 0)) *val = 0.0;
		else *val = inputArr[blockIdx.x * blockDim.x + threadIdx.x + inputOffset];
		break;
	default:
		*val = inputArr[blockIdx.x * blockDim.x + threadIdx.x + inputOffset % inputN];
	}
}

__device__ void getInputArrayValueForIndexingScheme(int pos, double *inputArr, const int inputOffset, const int inputN, int scheme, double *val)
{
	switch (scheme)
	{
	case 0:
		if ((pos + inputOffset) >= inputN) *val = 0.0;
		else *val = inputArr[pos + inputOffset];
		break;
	default:
		*val = inputArr[(pos + inputOffset) % inputN];
	}
}

__device__ void getInputArrayValueForIndexingScheme(int pos, __int32 *inputArr, const int inputOffset, const int inputN, int scheme, __int32 *val)
{
	switch (scheme)
	{
	case 0:
		if ((pos + inputOffset) >= inputN) *val = 0.0;
		else *val = inputArr[pos + inputOffset];
		break;
	default:
		*val = inputArr[(pos + inputOffset) % inputN];
	}
}

/******************************************************************************************************************/
/* double to double kernel maps */
/******************************************************************************************************************/

/* Kernel for adding a constant to an array */
__global__ void _kernel_ddmapAddSubtract(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val + d;
	}
}
/* Kernel for adding two arrays */
__global__ void _kernel_ddmap2Add(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const ThreadBlocks inputN, double *outputArr)
{
	double val1, val2;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input1Arr, input1Offset, inputN.N, 0, &val1);
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input2Arr, input2Offset, inputN.N, 0, &val2);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val1 + val2;
	}
}
/* Kernel for subtracting a constant from an array */
__global__ void _kernel_ddmapSubtract2(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = d - val;
	}
}
/* Kernel for subtracting two arrays*/
__global__ void _kernel_ddmap2Subtract(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const ThreadBlocks inputN, double *outputArr)
{
	double val1, val2;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input1Arr, input1Offset, inputN.N, 0, &val1);
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input2Arr, input2Offset, inputN.N, 0, &val2);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val1 - val2;
	}
}
/* Kernel for multiplying an array by a constant */
__global__ void _kernel_ddmapMultiply(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val * d;
	}
}
/* Kernel for multiplying two arrays */
__global__ void _kernel_ddmap2Multiply(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const ThreadBlocks inputN, double *outputArr)
{
	double val1, val2;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input1Arr, input1Offset, inputN.N, 0, &val1);
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input2Arr, input2Offset, inputN.N, 0, &val2);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val1 * val2;
	}
}
/* Kernel for dividing an array by a constant */
__global__ void _kernel_ddmapDivide(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val / d;
	}
}
/* Kernel for dividing a constant by an array */
__global__ void _kernel_ddmapDivide2(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = d / val;
	}
}
/* Kernel for dividing two arrays*/
__global__ void _kernel_ddmap2Divide(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const ThreadBlocks inputN, double *outputArr)
{
	double val1, val2;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input1Arr, input1Offset, inputN.N, 0, &val1);
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input2Arr, input2Offset, inputN.N, 0, &val2);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val1 / val2;
	}
}
/* Kernel for raising an array to the power of a constant*/
__global__ void _kernel_ddmapPower(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = pow(val, d);
	}
}
/* Kernel for raising a constant to the power of each array element */
__global__ void _kernel_ddmapPower2(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = pow(d, val);
	}
}
/* Kernel for raising each element of one array to the power of one element in another */
__global__ void _kernel_ddmap2Power(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const ThreadBlocks inputN, double *outputArr)
{
	double val1, val2;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input1Arr, input1Offset, inputN.N, 0, &val1);
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input2Arr, input2Offset, inputN.N, 0, &val2);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = pow(val1, val2);
	}
}
/* Kernel for square rooting an array */
__global__ void _kernel_ddmapSqrt(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = sqrt(val);
	}
}
/* Kernel for inverse cos of each element of an array */
__global__ void _kernel_ddmapArcCos(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = acos(val);
	}
}
/* Kernel for cos of each element of an array */
__global__ void _kernel_ddmapCos(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = cos(val);
	}
}
/* Kernel for hyperbolic cos of each element of an array */
__global__ void _kernel_ddmapCosh(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = cosh(val);
	}
}

__global__ void _kernel_ddmapArcSin(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = asin(val);
	}
}

__global__ void _kernel_ddmapSin(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = sin(val);
	}
}

__global__ void _kernel_ddmapSinh(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = sinh(val);
	}
}

__global__ void _kernel_ddmapArcTan(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = atan(val);
	}
}

__global__ void _kernel_ddmapTan(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = tan(val);
	}
}

__global__ void _kernel_ddmapTanh(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = tanh(val);
	}
}

__global__ void _kernel_ddmapLog(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = log(val);
	}
}

__global__ void _kernel_ddmapLog10(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = log10(val);
	}
}

__global__ void _kernel_ddmapExp(double *inputArr, const int inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = exp(val);
	}
}

/******************************************************************************************************************/
/* double to bool kernel maps */
/******************************************************************************************************************/

/* Kernel for calculating elementwise greater than value over constant and array */
__global__ void _kernel_dbmapGT(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, __int32 *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val > d;
	}
}

/* Kernel for calculating elementwise greater than value over array and constant */
__global__ void _kernel_dbmapGT2(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, __int32 *outputArr)
{

	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = d > val;
	}
}

/* Kernel for calculating elementwise greater than value over two arrays */
__global__ void _kernel_dbmap2GT(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const ThreadBlocks inputN, __int32 *outputArr)
{
	double val1, val2;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input1Arr, input1Offset, inputN.N, 0, &val1);
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input2Arr, input2Offset, inputN.N, 0, &val2);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val1 > val2;
	}
}

/* Kernel for calculating elementwise greater than or equal value over constant and array */
__global__ void _kernel_dbmapGTE(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, __int32 *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val >= d;
	}
}

/* Kernel for calculating elementwise greater than or equal value over array and constant */
__global__ void _kernel_dbmapGTE2(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, __int32 *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = d >= val;
	}
}


/* Kernel for calculating elementwise greater than or equal value over two arrays */
__global__ void _kernel_dbmap2GTE(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const ThreadBlocks inputN, __int32 *outputArr)
{
	double val1, val2;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input1Arr, input1Offset, inputN.N, 0, &val1);
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input2Arr, input2Offset, inputN.N, 0, &val2);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val1 >= val2;
	}
}

/* Kernel for calculating elementwise less than value over array and constant */
__global__ void _kernel_dbmapLT(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, __int32 *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val < d;
	}
}

/* Kernel for calculating elementwise less than value over array and constant */
__global__ void _kernel_dbmapLT2(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, __int32 *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = d < val;
	}
}

/* Kernel for calculating elementwise less than value over two arrays */
__global__ void _kernel_dbmap2LT(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const ThreadBlocks inputN, __int32 *outputArr)
{
	double val1, val2;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input1Arr, input1Offset, inputN.N, 0, &val1);
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input2Arr, input2Offset, inputN.N, 0, &val2);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val1 < val2;
	}
	
}

/* Kernel for calculating elementwise less than or equal value over constant and array */
__global__ void _kernel_dbmapLTE(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, __int32 *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val <= d;
	}
}

/* Kernel for calculating elementwise less than or equal value over array and constant */
__global__ void _kernel_dbmapLTE2(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, __int32 *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = d <= val;
	}
}

/* Kernel for calculating elementwise less than or equal value over two arrays */
__global__ void _kernel_dbmap2LTE(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const ThreadBlocks inputN, __int32 *outputArr)
{
	double val1, val2;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input1Arr, input1Offset, inputN.N, 0, &val1);
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input2Arr, input2Offset, inputN.N, 0, &val2);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val1 <= val2;
	}
}

/* Kernel for calculating elementwise equality between array and constant */
__global__ void _kernel_dbmapEquality(double *inputArr, const int inputOffset, const ThreadBlocks inputN, const double d, __int32 *outputArr, const bool not)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = (val == d) ^ not;
	}
}

/* Kernel for calculating elementwise equality over two arrays */
__global__ void _kernel_dbmap2Equality(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const ThreadBlocks inputN, __int32 *outputArr, const bool not)
{
	double val1, val2;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input1Arr, input1Offset, inputN.N, 0, &val1);
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input2Arr, input2Offset, inputN.N, 0, &val2);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = (val1 == val2) ^ not;
	}
}

/******************************************************************************************************************/
/* bool to bool kernel maps */
/******************************************************************************************************************/

/* Kernel for calculating elementwise conditional AND between array and constant */
__global__ void _kernel_bbmapConditionalAnd(__int32 *inputArr, const int inputOffset, const ThreadBlocks inputN, const int d, __int32 *outputArr)
{
	__int32 val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val && d;
	}
}

/* Kernel for calculating elementwise conditional AND over two arrays */
__global__ void _kernel_bbmap2ConditionalAnd(__int32 *input1Arr, const int input1Offset, __int32 *input2Arr, const int input2Offset, const ThreadBlocks inputN, __int32 *outputArr)
{
	__int32 val1, val2;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input1Arr, input1Offset, inputN.N, 0, &val1);
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input2Arr, input2Offset, inputN.N, 0, &val2);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val1 && val2;
	}
}

/* Kernel for calculating elementwise conditional AND between array and constant */
__global__ void _kernel_bbmapConditionalOr(__int32 *inputArr, const int inputOffset, const ThreadBlocks inputN, const int d, __int32 *outputArr)
{
	__int32 val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val || d;
	}
}

/* Kernel for calculating elementwise conditional AND over two arrays */
__global__ void _kernel_bbmap2ConditionalOr(__int32 *input1Arr, const int input1Offset, __int32 *input2Arr, const int input2Offset, const ThreadBlocks inputN, __int32 *outputArr)
{
	__int32 val1, val2;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input1Arr, input1Offset, inputN.N, 0, &val1);
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, input2Arr, input2Offset, inputN.N, 0, &val2);
		outputArr[i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x] = val1 || val2;
	}
}

/******************************************************************************************************************/
/* double kernel reductions */
/******************************************************************************************************************/

/* Reduce to half the size */
__global__ void _kernel_ddreduceToHalf(double *inputArr, const __int32 inputOffset, const ThreadBlocks inputN, double *outputArr)
{
	double val;
	for (int i = 0; i < inputN.loopCount; ++i)
	{
		getInputArrayValueForIndexingScheme(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x, inputArr, inputOffset, inputN.N, 0, &val);
		if ((i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x) % 2 == 0)
			outputArr[(i*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x) / 2] = val;
	}
}

/******************************************************************************************************************/
/* __int32 kernel reductions */
/******************************************************************************************************************/



__global__ void _kernel_iiprefixSum(__int32 *inputArr, int n, __int32 *outputArr)
{
	extern __shared__ __int32 t1[];
	__int32 offset = 1;
	t1[2 * threadIdx.x] = inputArr[2 * threadIdx.x];
	t1[2 * threadIdx.x + 1] = inputArr[2 * threadIdx.x + 1];

	for(int	d = n>>1; d > 0; d >>= 1)	// build sum in place up the tree
	{
		__syncthreads();
		if (threadIdx.x < d)
		{
			int	ai = offset*(2 * threadIdx.x + 1) - 1;
			int	bi = offset*(2 * threadIdx.x + 2) - 1;
			t1[bi] += t1[ai];
		}
		offset *= 2;
	}

	if (threadIdx.x == 0) t1[n - 1] = 0; // clear the last element

	for(int	d = 1; d < n; d *= 2)// traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadIdx.x < d)
		{
			int	ai = offset*(2 * threadIdx.x + 1) - 1;
			int	bi = offset*(2 * threadIdx.x + 2) - 1;
			__int32 t = t1[ai];
			t1[ai] = t1[bi];
			t1[bi] += t;
		}
	}
	__syncthreads();
	outputArr[2 * threadIdx.x] = t1[2 * threadIdx.x];
	outputArr[2 * threadIdx.x + 1] = t1[2 * threadIdx.x + 1];
	//printf("%d %d ", outputArr[2 * threadIdx.x], outputArr[2 * threadIdx.x + 1]);

}

/******************************************************************************************************************/
/* double filters */
/******************************************************************************************************************/

/* Kernel for filtering double array based on boolean array predicate */
__global__ void _kernel_ddfilter(double *inputArr, __int32 *predicateArr, const ThreadBlocks inputN, __int32 *nres, double *outputArr)
{
	__shared__ __int32 l_n;

	for (int iter = 0; iter < inputN.loopCount; ++iter) {
		// zero the counter
		if (threadIdx.x == 0) l_n = 0;
		__syncthreads();

		// get the values of the array and the predicate
		double d;
		__int32 b, pos;

		//__int32 i = iter*inputN.thrBlockCount + blockIdx.x * blockDim.x + threadIdx.x;
		int i = (blockIdx.x * inputN.loopCount * inputN.threadCount) + iter * inputN.threadCount + threadIdx.x;

		if (i < inputN.N) {
			d = inputArr[i];
			b = predicateArr[i];
			if (b != 0)
				pos = atomicAdd(&l_n, 1); // increment the counter for those which are true
		}
		__syncthreads();

		// leader increments the global counter
		if (threadIdx.x == 0)
			l_n = atomicAdd(nres, l_n);
		__syncthreads();

		// threads with true predicates write their elements
		if (i < inputN.N && b != 0) {
			pos += l_n; // increment local pos by global counter
			outputArr[pos] = d;
		}
		__syncthreads();
	}
}

/* Kernel for filtering double array based a prefix counter */
__global__ void _kernel_ddfilterPrefix(double *inputArr, __int32 *prefixArr, const ThreadBlocks inputN, double *outputArr)
{
	for (int iter = 0; iter < inputN.loopCount; ++iter) {
		int i = (blockIdx.x * inputN.loopCount * inputN.threadCount) + iter * inputN.threadCount + threadIdx.x;
		if (prefixArr[i] > 0 && i < inputN.N) 
		{
			if (i == 0 || prefixArr[i - 1] < prefixArr[i]) 
			{
				outputArr[prefixArr[i]-1] = inputArr[i-1];
			}
		}
	}
}

/* Kernel for filtering double array based a prefix counter */
__global__ void _kernel_iiInit(__int32 *prefixArr, const ThreadBlocks inputN, __int32 val)
{
	for (int iter = 0; iter < inputN.loopCount; ++iter) {
		int i = (blockIdx.x * inputN.loopCount * inputN.threadCount) + iter * inputN.threadCount + threadIdx.x;
		if (i < inputN.N)
		{
			prefixArr[i] = val;
		}
	}
}

/******************************************************************************************************************/
/* double to double maps */
/******************************************************************************************************************/
typedef double(*dbl_func)(double, double);
__device__ dbl_func add_kernel = _kernel_add<double, double>;
__device__ dbl_func subtract_kernel = _kernel_subtract<double, double>;
__device__ dbl_func multiply_kernel = _kernel_multiply<double, double>;
__device__ dbl_func divide_kernel = _kernel_divide<double, double>;


int ddmapAdd(double *inputArr, const int inputOffset, const int inputN, const double d, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	dbl_func add_kernel_h;
	hipMemcpyFromSymbol(&add_kernel_h, HIP_SYMBOL(add_kernel), sizeof(dbl_func));
	_kernel_map_op<double, double> << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr, add_kernel_h);
	return hipGetLastError();
}

int ddmap2Add(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	dbl_func add_kernel_h;
	hipMemcpyFromSymbol(&add_kernel_h, HIP_SYMBOL(add_kernel), sizeof(dbl_func));
	_kernel_map2_op<double, double> << < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr, add_kernel_h);
	return hipGetLastError();
}

int ddmapSubtract(double *inputArr, const int inputOffset, const int inputN, const double d, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	dbl_func subtract_kernel_h;
	hipMemcpyFromSymbol(&subtract_kernel_h, HIP_SYMBOL(subtract_kernel), sizeof(dbl_func));
	_kernel_map_op<double, double> << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr, subtract_kernel_h);
	return hipGetLastError();
}

int ddmapSubtract2(double *inputArr, const int inputOffset, const int inputN, const double d, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	dbl_func subtract_kernel_h;
	hipMemcpyFromSymbol(&subtract_kernel_h, HIP_SYMBOL(subtract_kernel), sizeof(dbl_func));
	_kernel_map_op2<double, double> << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr, subtract_kernel_h);
	return hipGetLastError();
}

int ddmap2Subtract(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	dbl_func subtract_kernel_h;
	hipMemcpyFromSymbol(&subtract_kernel_h, HIP_SYMBOL(subtract_kernel), sizeof(dbl_func));
	_kernel_map2_op<double, double> << < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr, subtract_kernel_h);
	return hipGetLastError();
}

int ddmapMultiply(double *inputArr, const int inputOffset, const int inputN, const double d, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	dbl_func multiply_kernel_h;
	hipMemcpyFromSymbol(&multiply_kernel_h, HIP_SYMBOL(multiply_kernel), sizeof(dbl_func));
	_kernel_map_op<double, double> << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr, multiply_kernel_h);
	return hipGetLastError();
}

int ddmap2Multiply(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	dbl_func multiply_kernel_h;
	hipMemcpyFromSymbol(&multiply_kernel_h, HIP_SYMBOL(multiply_kernel), sizeof(dbl_func));
	_kernel_map2_op<double, double> << < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr, multiply_kernel_h);
	return hipGetLastError();
}

int ddmapDivide(double *inputArr, const int inputOffset, const int inputN, const double d, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	dbl_func divide_kernel_h;
	hipMemcpyFromSymbol(&divide_kernel_h, HIP_SYMBOL(divide_kernel), sizeof(dbl_func));
	_kernel_map_op<double, double> << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr, divide_kernel_h);
	return hipGetLastError();
}

int ddmapDivide2(double *inputArr, const int inputOffset, const int inputN, const double d, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	dbl_func divide_kernel_h;
	hipMemcpyFromSymbol(&divide_kernel_h, HIP_SYMBOL(divide_kernel), sizeof(dbl_func));
	_kernel_map_op2<double, double> << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr, divide_kernel_h);
	return hipGetLastError();
}

int ddmap2Divide(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	dbl_func divide_kernel_h;
	hipMemcpyFromSymbol(&divide_kernel_h, HIP_SYMBOL(divide_kernel), sizeof(dbl_func));
	_kernel_map2_op<double, double> << < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr, divide_kernel_h);
	return hipGetLastError();
}

__int32 ddmapPower(double *inputArr, const int inputOffset, const int inputN, const double d, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapPower << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr);
	return hipGetLastError();
}

int ddmapPower2(double *inputArr, const int inputOffset, const int inputN, const double d, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapPower2 << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr);
	return hipGetLastError();
}

int ddmap2Power(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmap2Power << < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr);
	return hipGetLastError();
}

int ddmapSqrt(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapSqrt << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

__int32 ddmapArcCos(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapArcCos << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

int ddmapCos(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapCos << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

int ddmapCosh(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapCosh << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

int ddmapArcSin(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapArcSin << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

int ddmapSin(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapSin << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

int ddmapSinh(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapSinh << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise atan over an array */
int mapArcTan(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapArcTan << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise tan over an array */
int ddmapTan(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapTan << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise tanh over an array */
int ddmapTanh(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapTanh << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise log over an array */
int ddmapLog(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapLog << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise log10 over an array */
int ddmapLog10(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddmapLog10 << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

/******************************************************************************************************************/
/* double to bool maps */
/******************************************************************************************************************/

/* Function for calculating elementwise greater than value over array and constant */
int dbmapGT(double *inputArr, const int inputOffset, const int inputN, const double d, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmapGT << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise greater than value over array and constant */
int dbmapGT2(double *inputArr, const int inputOffset, const int inputN, const double d, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmapGT2 << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise greater than value over two arrays */
int dbmap2GT(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const int inputN, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmap2GT << < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise greater than or equal value over array and constant */
int dbmapGTE(double *inputArr, const int inputOffset, const int inputN, const double d, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmapGTE << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise greater than or equal value over array and constant */
int dbmapGTE2(double *inputArr, const int inputOffset, const int inputN, const double d, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmapGTE2 << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise greater than or equal over two arrays */
int dbmap2GTE(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const int inputN, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmap2GTE << < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise less than value over array and constant */
int dbmapLT(double *inputArr, const int inputOffset, const int inputN, const double d, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmapLT << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise less than value over array and constant */
int dbmapLT2(double *inputArr, const int inputOffset, const int inputN, const double d, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmapLT2 << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise less then value over two arrays */
int dbmap2LT(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const int inputN, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmap2LT << < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise less than or equal over array and constant */
__int32 dbmapLTE(double *inputArr, const int inputOffset, const int inputN, const double d, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmapLTE << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise less than or equal over array and constant */
int dbmapLTE2(double *inputArr, const int inputOffset, const int inputN, const double d, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmapLTE2 << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise less then or equal over two arrays */
int dbmap2LTE(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const int inputN, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmap2LTE << < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise equality over array and constant */
int dbmapEquality(double *inputArr, const int inputOffset, const int inputN, const double d, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmapEquality << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr, false);
	return hipGetLastError();
}

/* Function for calculating elementwise equality over two arrays */
int dbmap2Equality(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const int inputN, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmap2Equality << < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr, false);
	return hipGetLastError();
}

/* Function for calculating elementwise not equality over array and constant */
int dbmapNotEquality(double *inputArr, const int inputOffset, const int inputN, const double d, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmapEquality << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr, true);
	return hipGetLastError();
}

/* Function for calculating elementwise not equality over two arrays */
int dbmap2NotEquality(double *input1Arr, const int input1Offset, double *input2Arr, const int input2Offset, const __int32 inputN, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_dbmap2Equality << < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr, true);
	return hipGetLastError();
}

/******************************************************************************************************************/
/* bool to bool kernel maps */
/******************************************************************************************************************/

/* Function for calculating elementwise conditional AND over array and constant */
int bbmapConditionAnd(__int32 *inputArr, const int inputOffset, const int inputN, const double d, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_bbmapConditionalAnd << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise conditional AND over two arrays */
int bbmap2ConditionAnd(__int32 *input1Arr, const int input1Offset, __int32 *input2Arr, const int input2Offset, const __int32 inputN, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_bbmap2ConditionalAnd << < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise conditional OR over array and constant */
int bbmapConditionOr(__int32 *inputArr, const int inputOffset, const int inputN, const double d, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_bbmapConditionalOr << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, d, outputArr);
	return hipGetLastError();
}

/* Function for calculating elementwise conditional OR over two arrays */
int bbmap2ConditionOr(__int32 *input1Arr, const int input1Offset, __int32 *input2Arr, const int input2Offset, const __int32 inputN, __int32 *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_bbmap2ConditionalOr<< < tb.blockCount, tb.threadCount >> >(input1Arr, input1Offset, input2Arr, input2Offset, tb, outputArr);
	return hipGetLastError();
}



/******************************************************************************************************************/
/* double reductions */
/******************************************************************************************************************/

int ddreduceToHalf(double *inputArr, const int inputOffset, const int inputN, double *outputArr)
{
	ThreadBlocks tb = getThreadsAndBlocks(inputN);
	_kernel_ddreduceToHalf << < tb.blockCount, tb.threadCount >> >(inputArr, inputOffset, tb, outputArr);
	return hipGetLastError();
}

/******************************************************************************************************************/
/* double filters */
/******************************************************************************************************************/

/* Function for filtering a double array by a boolean array predicate */
int ddfilter(double *inputArr, __int32 *predicateArr, const int inputN, double *outputArr, __int32 *outputN)
{
	__int32 *prefixSum;
	int nP1 = inputN + 1; // prefix sum is one longer than array because of leading -1
	ThreadBlocks tb = getThreadsAndBlocks(nP1);
	
	hipMalloc(&prefixSum, nP1 * sizeof(int));
	// Calculate parallel prefix sum
	ScanBlockAllocation sba = preallocBlockSums(nP1);
	prescanArray(prefixSum, predicateArr, nP1, sba);
	deallocBlockSums(sba);
	// filter using prefix sum
	_kernel_ddfilterPrefix << < tb.blockCount, tb.threadCount >> >(inputArr, prefixSum, tb, outputArr);
	// copy length of array
	hipMemcpy(outputN, prefixSum + (inputN), sizeof(int), hipMemcpyDeviceToHost);
	// cleanup
	hipFree(prefixSum);
	return hipGetLastError();
}